
#include <hip/hip_runtime.h>
#include <stdio.h>

// Device code
__global__ void VecAdd(float* A, float* B, float* C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
};

// Host code
int main(int argc, char **argv) {
    int N = 1024;
    size_t size = N * sizeof(float);
    // Allocate input vectors h_A and h_B in host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);
    // Initialize input vectors
    for (int it=0; it<N; it++) {
        h_A[it] = it;
        h_B[it] = -it;
        h_C[it] = 2;
    };
    // Allocate vectors in device memory
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);
    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    // Invoke kernel
    int threadsPerBlock = 256;
	int blocksPerGrid = N / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    float res = 0.0;
    for (int it=0; it<N; it++) {
        res += h_C[it];
    };
    printf("%g\n", res);
    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    // Free host memory
	free(h_C);
	free(h_B);
    free(h_A);
};
